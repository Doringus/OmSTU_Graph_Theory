#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/generate.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

////////// Global variables section //////////

__constant__ int BLOCK_COUNT;
__constant__ int THREADS_PER_BLOCK;


////////// End section //////////

////////// Device functions section //////////

__device__
int getThreadId() {
    int blockId  = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}

__device__
int getBlockId() {
    return blockIdx.y * gridDim.x + blockIdx.x;
}

__device__
int get1dIndex(int row, int col, int width) {
    return width * row + col;
}

__device__
int takeParent(double* fitness, int startIndex, double prob) {
    int i;
    double sum = 0;
    for(i = startIndex; i < startIndex + THREADS_PER_BLOCK; ++i) {
        sum += fitness[i];
        if(prob < sum) {
            return i;
        }
    }
    return --i;
}

////////// End section //////////




////////// Kernels section //////////

__global__
void initRandomKernel(unsigned int seed, hiprandState_t* states) {
    int id = getThreadId();
    hiprand_init(seed, id, 0, &states[id]);
}

__global__
void uniformRandomSelectionKernel(hiprandState_t* states, double* selectionRandomArray) {
    selectionRandomArray[getThreadId()] = hiprand_uniform(&states[getThreadId()]);
}

__global__
void crossoverRandomPointsKernel(hiprandState_t* states, int* crossoverPointsArray, int max) {
    crossoverPointsArray[getThreadId()] = hiprand(&states[getThreadId()]) % (max - 1);
}

///
/// \brief fitnessKernel calculates fit for each individual (thread per individual)
/// \param matrix - graph matrix
/// \param population
/// \param fitness - array for result of calculation
/// \param populationSize
/// \param matrixSize - number of rows(or columns) in matrix
///
__global__
void fitnessKernel(int* matrix, int* population, double* fitness, int populationSize, int matrixSize) {
    double fit = 0;
    int startIndex = getThreadId() * matrixSize;
    for(int i = startIndex; i < startIndex + matrixSize - 1; ++i) {
        fit += matrix[get1dIndex(population[i], population[i + 1], matrixSize)];
    }
    fit += matrix[get1dIndex(population[startIndex + matrixSize - 1], population[startIndex], matrixSize)];
    fit = 1.0 / fit;
    fitness[getThreadId()] = fit;
    __syncthreads();
    // normilize fitness to roulette selection
    // calculate sum of fitness in block
    double sum = 0;
    startIndex = getBlockId() * THREADS_PER_BLOCK;
    for(int i = startIndex; i < startIndex + THREADS_PER_BLOCK; ++i) {
        sum += fitness[i];
    }
    fit = fit / sum;

//    printf("%f %d\n", fit, getThreadId());
    __syncthreads();
    fitness[getThreadId()] = fit;
}

///
/// \brief breedKernel makes roulette selection, and perfroms OX crossover. Thread per two individuals
/// \param population
/// \param nextGeneration
/// \param fitness
/// \param probability - array with random numbers for roulette selection
/// \param crossoverPoints - array with random numbers for crossover
/// \param size - genes count in individual
///
__global__
void breedKernel(int* population, int* nextGeneration, double* fitness, double* probability, int* crossoverPoints, int size) {
    // thread runs roulette twice and finds two parents and starts crossover

    int startIndex = getBlockId() * THREADS_PER_BLOCK;
    int parent1Index = takeParent(fitness, startIndex, probability[getThreadId() * 2]);
    int parent2Index = takeParent(fitness, startIndex, probability[getThreadId() * 2 + 1]);
    // start ox crossover
    int firstPoint = crossoverPoints[getThreadId() * 2];
    int secondPoint = crossoverPoints[getThreadId() * 2 + 1];
    if(firstPoint > secondPoint) {
        int tmp = firstPoint;
        firstPoint = secondPoint;
        secondPoint = tmp;
    }
    for(int i = firstPoint; i < secondPoint; ++i) {
        // child 1
        nextGeneration[get1dIndex(getThreadId() * 2, i, size)] = population[get1dIndex(parent1Index, i, size)];
        // child 2
        nextGeneration[get1dIndex(getThreadId() * 2 + 1, i, size)] = population[get1dIndex(parent2Index, i, size)];
    }
    int placeIndex1 = secondPoint;
    int placeIndex2 = secondPoint;
    for(int i = 0; i < size; ++i) {
        int index = (secondPoint + i) % size;
        int parent1Gene = population[get1dIndex(parent1Index, index, size)];
        int parent2Gene = population[get1dIndex(parent2Index, index, size)];
        bool isInSlice = false;
        for(int j = firstPoint; j < secondPoint; ++j) {
            if(population[get1dIndex(parent1Index, j, size)] == parent2Gene) {
                isInSlice = true;
                break;
            }
        }
        if(!isInSlice) {
            nextGeneration[get1dIndex(getThreadId() * 2, placeIndex1 % size, size)] = parent2Gene;
            placeIndex1++;
        }
        isInSlice = false;
        for(int j = firstPoint; j < secondPoint; ++j) {
            if(population[get1dIndex(parent2Index, j, size)] == parent1Gene) {
                isInSlice = true;
                break;
            }
        }
        if(!isInSlice) {
            nextGeneration[get1dIndex(getThreadId() * 2 + 1, placeIndex2 % size, size)] = parent1Gene;
            placeIndex2++;
        }
    }
}

__global__
void mutationKernel(int* population, double* prob, int* randomPoints, int size) {
    int index = getThreadId();
    if(prob[index] < 0.2f) {
        // swap mutation;
        int tmp = population[get1dIndex(index, randomPoints[index * 2], size)];
        population[get1dIndex(index, randomPoints[index * 2], size)] = population[get1dIndex(index, randomPoints[index * 2 + 1], size)];
        population[get1dIndex(index, randomPoints[index * 2 + 1], size)] = tmp;
    }
}

__global__
void copyToMigrationPoolKernel(int* population, int* pool, int genesPerThread, int size) {
    int index = getBlockId() * THREADS_PER_BLOCK * size + threadIdx.x * genesPerThread;
    for(int i = 0; i < genesPerThread; ++i) {
        pool[getThreadId() + i] = population[index];
        ++index;
    }
}

__global__
void copyFromMigrationPoolKernel(int* population, int* pool, int genesPerThread, int size) {
    int to = ((getBlockId() + 1) % BLOCK_COUNT) * THREADS_PER_BLOCK * size + threadIdx.x * genesPerThread;
    for(int i = 0; i < genesPerThread; ++i) {
        population[to] = pool[getThreadId() + i];
        ++to;
    }
}

////////// End section //////////

extern "C"
int* gaCuda(int* graphMatrix, int matrixSize, int* population, int populationPerIslandSize, int populationSize) {
    // Cuda configuration
    int threadsPerBlock = populationPerIslandSize;
    int numBlocks = 4;
    int migrationPoolPerIsland = 4;
    // Thread per gene
    int threadsNumberForMigration = migrationPoolPerIsland * matrixSize;
    // best individual
    int* best = (int*)malloc(matrixSize * sizeof(int));

    int* population1d;
    int* nextPopulation1d;
    int* migrationPool;
    double* fitnessArray;
    double* selectionRandomArray;
    int* randomPointsArray; // points for crossover and mutation
    int* matrixArray;
    hiprandState_t* states;
    // Allocate memory
    hipMalloc(&matrixArray, matrixSize * matrixSize * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&population1d, populationSize * matrixSize * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&nextPopulation1d, populationSize * matrixSize * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&migrationPool, migrationPoolPerIsland * numBlocks * matrixSize * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&fitnessArray, populationSize * sizeof(double));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&states, populationPerIslandSize * 2 * numBlocks * sizeof(hiprandState_t));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&selectionRandomArray, populationPerIslandSize * numBlocks * sizeof(double));
    gpuErrchk(hipPeekAtLastError());
    hipMalloc(&randomPointsArray, populationPerIslandSize * 2 * numBlocks * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    // Fill memory
    hipMemcpy(matrixArray, graphMatrix, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpy(population1d, population, populationSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpyToSymbol(HIP_SYMBOL(BLOCK_COUNT), &numBlocks, sizeof(int), 0, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    hipMemcpyToSymbol(HIP_SYMBOL(THREADS_PER_BLOCK), &threadsPerBlock, sizeof(int), 0, hipMemcpyHostToDevice);
    gpuErrchk(hipPeekAtLastError());
    hipMemset(nextPopulation1d, -1, populationSize * matrixSize * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    fitnessKernel<<<numBlocks, threadsPerBlock>>>(matrixArray, population1d, fitnessArray, populationPerIslandSize, matrixSize);
    for(int i = 0; i < 2000; ++i) {
        initRandomKernel<<<numBlocks, threadsPerBlock>>>(time(0), states);
        gpuErrchk(hipPeekAtLastError());
        uniformRandomSelectionKernel<<<numBlocks, threadsPerBlock>>>(states, selectionRandomArray);
        gpuErrchk(hipPeekAtLastError());
        crossoverRandomPointsKernel<<<numBlocks, threadsPerBlock>>>(states, randomPointsArray, matrixSize);
        gpuErrchk(hipPeekAtLastError());
        breedKernel<<<numBlocks, threadsPerBlock / 2>>>(population1d, nextPopulation1d, fitnessArray, selectionRandomArray, randomPointsArray, matrixSize);
        mutationKernel<<<numBlocks, threadsPerBlock>>>(nextPopulation1d, selectionRandomArray, randomPointsArray, matrixSize);
        gpuErrchk(hipPeekAtLastError());
        fitnessKernel<<<numBlocks, threadsPerBlock>>>(matrixArray, nextPopulation1d, fitnessArray, populationPerIslandSize, matrixSize);
        gpuErrchk(hipPeekAtLastError());
        if(i % 100 == 0 && i != 0) {
            copyToMigrationPoolKernel<<<numBlocks, threadsNumberForMigration>>>(population1d, migrationPool, 1, matrixSize);
            gpuErrchk(hipPeekAtLastError());
            copyFromMigrationPoolKernel<<<numBlocks, threadsNumberForMigration>>>(population1d, migrationPool, 1, matrixSize);
            gpuErrchk(hipPeekAtLastError());
        }
        hipDeviceSynchronize();
        hipMemcpy(population1d, nextPopulation1d,  populationSize * matrixSize * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemset(nextPopulation1d, -1, populationSize * matrixSize * sizeof(int));
        gpuErrchk(hipPeekAtLastError());
    }
    int index = thrust::max_element(thrust::device, fitnessArray, fitnessArray + populationSize) - fitnessArray;
    hipMemcpy(best, population1d + index * matrixSize, matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();
    hipFree(population1d);
    hipFree(nextPopulation1d);
    hipFree(fitnessArray);
    hipFree(matrixArray);
    hipFree(states);
    hipFree(randomPointsArray);
    hipFree(selectionRandomArray);
    hipFree(migrationPool);
    return best;
}
